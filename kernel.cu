/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 16

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     * Use shared memory for tiling
     *
     ********************************************************************/

    // INSERT KERNEL CODE HERE

    __shared__ float ds_A[TILE_SIZE][TILE_SIZE];
    __shared__ float ds_B[TILE_SIZE][TILE_SIZE];

    int bx = blockIdx.x; int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;

    int Row = by*blockDim.y + ty;
    int Col = bx*blockDim.x + tx;

    float Pvalue = 0.0;
    int p;
    for (p = 0; p < (k-1) / TILE_SIZE + 1; ++p) {
        if(Row < m && p * TILE_SIZE+tx < k) {
            ds_A[ty][tx] = A[Row * k + p * TILE_SIZE + tx];
        } else {
            ds_A[ty][tx] = 0.0;
        }
        if (p*TILE_SIZE+ty < k && Col < n) {
            ds_B[ty][tx] = B[(p*TILE_SIZE + ty) * n + Col];
        } else {
            ds_B[ty][tx] = 0.0;
        }
        __syncthreads();
        if(Row < m && Col < n) {
            int i;
            for (i = 0; i < TILE_SIZE; ++i) {
                Pvalue += ds_A[ty][i] * ds_B[i][tx];
            }
        }
        __syncthreads();
    } /* end of outer for loop */
    if (Row < m && Col < n)
        C[Row*n + Col] = Pvalue;
} /* end of kernel */



void basicSgemm(char transa, char transb, int m, int n, int k, float alpha, const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc)
{
    if ((transa != 'N') && (transa != 'n')) {
	printf("unsupported value of 'transa'\n");
    	return;
    }

    if ((transb != 'N') && (transb != 'n')) {
	printf("unsupported value of 'transb'\n");
	return;
    }

    if ((alpha - 1.0f > 1e-10) || (alpha - 1.0f < -1e-10)) {
	printf("unsupported value of alpha\n");
	return;
    }

    if ((beta - 0.0f > 1e-10) || (beta - 0.0f < -1e-10)) {
	printf("unsupported value of beta\n");
	return;
    }

    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = TILE_SIZE;

    //INSERT CODE HERE

    dim3 DimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 DimGrid((n-1)/BLOCK_SIZE+1, (m-1)/BLOCK_SIZE+1, 1);


    // Invoke CUDA kernel -----------------------------------------------------

    //INSERT CODE HERE
    mysgemm<<<DimGrid, DimBlock>>>(m, n, k, A, B, C);



}

__global__ void s_mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     ********************************************************************/

    // INSERT KERNEL CODE HERE
    int Row = blockIdx.y*blockDim.y + threadIdx.y;
    int Col = blockIdx.x*blockDim.x + threadIdx.x;

    if ((Row < m) && (Col < n)) {
        float Pvalue = 0;
        int kid;
        for (kid=0; kid < k; kid++) {
            Pvalue += A[Row*k+kid]*B[kid*n+Col];
        }
        C[Row*n+Col] = Pvalue;
        
    }
}

    void s_basicSgemm(char transa, char transb, int m, int n, int k, float alpha, const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc)
{
    if ((transa != 'N') && (transa != 'n')) {
    printf("unsupported value of 'transa'\n");
        return;
    }

    if ((transb != 'N') && (transb != 'n')) {
    printf("unsupported value of 'transb'\n");
    return;
    }

    if ((alpha - 1.0f > 1e-10) || (alpha - 1.0f < -1e-10)) {
    printf("unsupported value of alpha\n");
    return;
    }

    if ((beta - 0.0f > 1e-10) || (beta - 0.0f < -1e-10)) {
    printf("unsupported value of beta\n");
    return;
    }

    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = 16; // Use 16x16 thread blocks

    //INSERT CODE HERE
    dim3 DimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 DimGrid((n-1)/BLOCK_SIZE+1, (m-1)/BLOCK_SIZE+1, 1);



    // Invoke CUDA kernel -----------------------------------------------------

    //INSERT CODE HERE


    s_mysgemm<<<DimGrid, DimBlock>>>(m, n, k, A, B, C);

}



